#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int* c, const int* a, const int* b, int size) {
    int* dev_a = NULL;
    int* dev_b = NULL;
    int* dev_c = NULL;

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    // 2 is number of computational blocks and (size + 1) / 2 is a number of threads in a block
    addKernel<<<2, (size + 1) / 2>>>(dev_c, dev_a, dev_b, size);

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

int main(int argc, char** argv) {
    const int arraySize = 5;
    const int a[arraySize] = {  1,  2,  3,  4,  5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    const int c[arraySize] = {11, 12, 13, 14, 15};
    const int d[arraySize] = {100, 200, 300, 400, 500};

    int result[arraySize] = { 0 };
    int input = atoi(argv[1]);

    if (input == 1) {
    	addWithCuda(result, a, b, arraySize);
    }
    else {
    	addWithCuda(result, c, d, arraySize);
    }
    //Printing the output
    printf("Addition of two arrays = {%d, %d, %d, %d, %d}\n", result[0], result[1], result[2], result[3], result[4]);

    hipDeviceReset();

    return 0;
}