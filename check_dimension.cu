#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void checkIndex(void)
{
    printf("threadIdx:(%d ,%d ,%d)blockIdx:(%d , %d ,%d)blockDim:(%d ,%d ,%d)gridDim(%d,%d,%d)\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}
int main(int argc, char **argv)
{
    int nElem = 6;
    dim3 block(3);
    dim3 grid((nElem + block.x - 1) / block.x);
    checkIndex<<<grid, block>>>();
    hipDeviceSynchronize();
    return 0;
}